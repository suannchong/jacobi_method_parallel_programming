
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 10

__global__ void add(int*a, int*b, int*c){
	c[blockIdx.x] = a[blockIdx.x+1] + b[blockIdx.x];
	
	//*c = *a + *b;
}

int main(void){
	int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	int size = N*sizeof(int);

	// Allocate space for device copies of a, b, c
	hipMalloc((void**) &d_a, size); 
	hipMalloc((void**) &d_b, size);
	hipMalloc((void**) &d_c, size);

	// set up input values
	a = (int*) calloc(1,size); 
	b = (int*) calloc(1,size); 
	c = (int*) calloc(1,size);

	for (int i=0; i<N; i++){
		a[i] = i*1;
		b[i] = i*2;
		printf("a[%d]=%d \t b[%d]=%d\n",i,a[i],i,b[i]);
	
	}

	// copy inputs to device
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	// Launch add kernel on GPU
	add<<<N,1>>>(d_a, d_b, d_c);

	// copy result back to host
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
	
	for (int i=0; i<N; i++){	
		printf("c[%d] = %d\n", i, c[i]);
	}

	// Clean up
	free(a); free(b); free(c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);

	return 0;

	
}

