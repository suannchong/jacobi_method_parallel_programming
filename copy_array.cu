
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void kernel(double* d_array, double* d_sum){
	// int i = (blockIdx.x*blockDim.x) + threadIdx.x;
	// int x = i % 2560;
	// int y = i % 2560;

	for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < 2560*2560; i += blockDim.x*gridDim.x){
		d_array[i] = i+0.1;
		printf("d_array[%d] = %.2f\n", i ,d_array[i] );
		*d_sum += d_array[i];
	}
}

int main(int argc, char** argv){
	int N = 2560*2560;
	double* array = (double*) calloc(N,sizeof(double));
	double sum = 2.03;
	double* d_array, *d_sum;
	size_t size = N*sizeof(double);

	hipMalloc((void**) &d_array, size);
	hipMalloc((void**) &d_sum, sizeof(double));

	hipMemcpy(d_array,array,size,hipMemcpyHostToDevice);
	hipMemcpy(d_sum, &sum, sizeof(double), hipMemcpyHostToDevice);

	for (int k = 0; k < 100000; k++){
		kernel<<<1,1024>>>(d_array,d_sum);

		hipMemcpy(array, d_array, size, hipMemcpyDeviceToHost);
		hipMemcpy(&sum, d_sum, sizeof(double), hipMemcpyDeviceToHost);

		// for (int i=0; i<N; i++){
		// 	printf("array[%d] = %.2f\n",i, array[i] );
		// }

		printf("sum = %.2f\n", sum );
	}

	hipFree(d_array);
	hipFree(d_sum);
	return 0;
}